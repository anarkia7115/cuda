
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* a, int N)
{
        int i;
        for (i = 0; i < N; ++i)
	        a[i] = rand();
}

#define N 512
int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  
  a = (int *)malloc(size);
  random_ints(a, N);
  b = (int *)malloc(size);
  random_ints(b, N);
  c = (int *)malloc(size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  add<<<N,1>>>(d_a, d_b, d_c);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    std::cout << *(a + i) << "+" 
      << *(b + i) << "="
      << *(c + i) << std::endl;
  }
  std::cout << "hello" << std::endl;

  free(a);
  free(b);
  free(c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
